#include "hip/hip_runtime.h"
/** CUDA sample: adding two arrays with libgpuvm */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <signal.h>
#include <stdlib.h>
#include <unistd.h>

#include <sys/syscall.h>
#include <sys/types.h>

extern "C" {
  #include "../../../src/gpuvm.h"
}

// macros to check for errors
#define CHECK(x) \
	{\
		int res = (x);															\
		if(res != hipSuccess) {										\
			printf(#x "\n");													\
			printf("%d\n", res);											\
			exit(-1);																	\
		}																						\
	}

#define CHECK_NULL(x) \
	if(x == NULL) {\
	printf(#x "\n");\
	exit(-1);\
	}

#define COUNT 4

#define N (1024 * 13 + 64)
#define SZ (N * sizeof(int))
#define NRUNS 1

void __global__ add_arrays_kernel(int *c, int *a, int *b, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

/** adds arrays on GPU, by calling OpenCL kernel; note that device pointers
		which correspond to host pointers are obtained using gpuvm_xlate(), and are
		not passed separately to the function */
void add_arrays_on_gpu(int *c, int *a, int *b, int n) {
	CHECK(gpuvm_kernel_begin(a, 0, GPUVM_READ_WRITE));
	CHECK(gpuvm_kernel_begin(b, 0, GPUVM_READ_WRITE));
	CHECK(gpuvm_kernel_begin(c, 0, GPUVM_READ_WRITE));

	add_arrays_kernel<<<n/64, 64>>>
		((int*)gpuvm_xlate(c, 0), (int*)gpuvm_xlate(a, 0), (int*)gpuvm_xlate(b, 0), n);

	CHECK(hipDeviceSynchronize());

	// on kernel end
	//printf("actions on kernel end\n");
	CHECK(gpuvm_kernel_end(a, 0));
	CHECK(gpuvm_kernel_end(b, 0));
	CHECK(gpuvm_kernel_end(c, 0));
	// CHECK(hipMemcpy(a, gpuvm_xlate(a, 0), n * sizeof(int), hipMemcpyDeviceToHost));
	// CHECK(hipMemcpy(b, gpuvm_xlate(b, 0), n * sizeof(int), hipMemcpyDeviceToHost));
	// CHECK(hipMemcpy(c, gpuvm_xlate(c, 0), n * sizeof(int), hipMemcpyDeviceToHost));
	// CHECK(hipDeviceSynchronize());
}

int main(int argc, char** argv) {
	
	//CHECK(gpuvm_pre_init(GPUVM_THREADS_BEFORE_INIT));
	//CHECK(gpuvm_pre_init(GPUVM_THREADS_AFTER_INIT));

	// initialize GPUVM
	CHECK(gpuvm_init
				(1, 0, GPUVM_CUDA | GPUVM_UNLINK_NO_SYNC_BACK | GPUVM_WRITER_SIG_BLOCK));

	// allocate host data
	int *ha = 0, *hb = 0, *hc = 0, *hg = 0;
	ha = (int*)malloc(SZ);
	hb = (int*)malloc(SZ);
	hc = (int*)malloc(SZ);
	hg = (int*)malloc(SZ);
	CHECK_NULL(ha);
	CHECK_NULL(hb);
	CHECK_NULL(hg);
	for(int i = 0; i < N; i++) {
		ha[i] = i;
		hb[i] = i + 1;
	}

	// allocate device data
	int *da, *db, *dc;
	CHECK(hipMalloc(&da, SZ));
	CHECK(hipMalloc(&db, SZ));
	CHECK(hipMalloc(&dc, SZ));

	// link host buffers to device buffers
	//printf("linking buffers\n");
	CHECK(gpuvm_link(ha, SZ, 0, da, GPUVM_ON_HOST));
	CHECK(gpuvm_link(hb, SZ, 0, db, GPUVM_ON_HOST));
	CHECK(gpuvm_link(hc, SZ, 0, dc, GPUVM_ON_HOST));

	// before-kernel actions
	printf("adding arrays\n");

	unsigned irun;
	for(irun = 0; irun < NRUNS; irun++) {
		// do work on GPU
		add_arrays_on_gpu(hc, ha, hb, N);

		// evaluate "gold" result
		for(int i = 0; i < N; i++)
			hg[i] = ha[i] + hb[i];

		// check result
		for(int i = 0; i < N; i++) {
			if(hg[i] != hc[i]) {
				printf("check: FAILED\n");
				printf("hg[%d] != hc[%d]: %d != %d\n", i, i, hg[i], hc[i]);
				exit(-1);
			}
		}
		printf("check: PASSED\n");
	}  // for(irun)

	// print result
	printf("printing result\n");
	int step = 1536;
	for(int i = 0; i < N; i += step)
		printf("hc[%d] = %d\n", i, hc[i]);

	// unlink
	CHECK(gpuvm_unlink(ha, 0));
	CHECK(gpuvm_unlink(hb, 0));
	CHECK(gpuvm_unlink(hc, 0));

	// free CUDA device arrays
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	// free host memory
	free(ha);
	free(hb);
	free(hc);
	free(hg);

	return 0;
}  // end of main()
